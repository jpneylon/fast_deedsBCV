#include "hip/hip_runtime.h"
/*
 *       Written by Jack Neylon, PhD
 *                  University of California Los Angeles
 *                  200 Medical Plaza, Suite B265
 *                  Los Angeles, CA 90095
 *       2024-04-05
*/

#include "cuda_transformations_kernels.cuh"
#include "thrust/device_ptr.h"
#include "thrust/reduce.h"
#include <thrust/extrema.h>
#include <thrust/count.h>

#define THREADS 512

int3 cuda_find_grid( int sz1, dim3 block, hipDeviceProp_t devProp )
{
    int3 imSize;
    imSize.x = sz1 / block.x;
    imSize.y = 1;
    imSize.z = 1;
    if ( sz1 % block.x > 0 ) imSize.x++;
    if ( imSize.x > devProp.maxGridSize[1] )
    {
        imSize.y = imSize.x / devProp.maxGridSize[1];
        if ( imSize.x % devProp.maxGridSize[1] > 0 ) imSize.y++;
        imSize.x = devProp.maxGridSize[1];

        if ( imSize.y > devProp.maxGridSize[1] )
        {
            imSize.z = imSize.y / devProp.maxGridSize[1];
            if ( imSize.y % devProp.maxGridSize[1] > 0 ) imSize.z++;
            imSize.y = devProp.maxGridSize[1];
        }
    }
    return imSize;
}

extern "C" void
cuda_interp3( float* interp,
              float* input,
              float* x1,
              float* y1,
              float* z1,
              int m,int n,int o,
              int m2,int n2,int o2,
              bool flag)
{
    //hipDeviceReset();

    if (false)
    {
        size_t freeMem, totalMem;
        checkCudaErrors(hipMemGetInfo(&freeMem,&totalMem));
        printf("\n ||| CUDA_INTERP3 : Device - Initial Free Memory: %lu / %lu ||| \n",freeMem,totalMem);
    }

    hipDeviceProp_t devProp;
    hipGetDeviceProperties( &devProp, 0 );
    // dim3 block( devProp.maxThreadsPerBlock );
    dim3 block( THREADS );

    int sz1=m*n*o;
    int sz2=m2*n2*o2;
    int3 grid = cuda_find_grid( sz1, block, devProp );
    dim3 gridIM( grid.x, grid.y, grid.z );

    float *d_interp, *d_x1, *d_y1, *d_z1;
    float *d_input;

    checkCudaErrors( hipMalloc( (void**) &d_interp, sz1 * sizeof(float) ) );
	checkCudaErrors( hipMemset( d_interp, 0, sz1 * sizeof(float) ) );
    checkCudaErrors( hipMalloc( (void**) &d_input, sz2 * sizeof(float) ) );
	checkCudaErrors( hipMemcpy( d_input, input, sz2 * sizeof(float), hipMemcpyHostToDevice ) );

    ///////////////// Bind Inputs to 3D Texture Arrays //////////////////////////////////////////////
    // hipArray *d_input3DArray;
    // hipExtent inputExtent = make_hipExtent(m2, n2, o2);
    // hipChannelFormatDesc floatTex = hipCreateChannelDesc<float>();
    // checkCudaErrors(hipMalloc3DArray(&d_input3DArray, &floatTex, inputExtent));

	// hipMemcpy3DParms CopyParams = {0};
	// CopyParams.srcPtr	    =	make_hipPitchedPtr(input,inputExtent.width*sizeof(float), inputExtent.width, inputExtent.height);
	// CopyParams.dstArray	    =	d_input3DArray;
	// CopyParams.extent       =	inputExtent;
	// CopyParams.kind		    =	hipMemcpyHostToDevice;
	// hipMemcpy3D(&CopyParams);

    // hipResourceDesc resDesc;
    // memset(&resDesc, 0, sizeof(resDesc));
    // resDesc.resType = hipResourceTypeArray;
    // resDesc.res.array.array = d_input3DArray;

    // hipTextureDesc texDesc;
    // memset(&texDesc, 0, sizeof(hipTextureDesc));
    
	// texDesc.normalizedCoords	=	false;
	// texDesc.filterMode		    =	hipFilterModeLinear;
	// texDesc.addressMode[0]	    =	hipAddressModeClamp;
	// texDesc.addressMode[1]	    =	hipAddressModeClamp;
	// texDesc.addressMode[2]	    =	hipAddressModeClamp;
    // texDesc.readMode            =   hipReadModeElementType;

    // hipTextureObject_t texData;
	// checkCudaErrors(hipCreateTextureObject(&texData, &resDesc, &texDesc, NULL));

    checkCudaErrors(hipMalloc((void **)&d_x1 , sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc((void **)&d_y1 , sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc((void **)&d_z1 , sz1 * sizeof(float) ) );
    checkCudaErrors(hipMemcpy(d_x1, x1, sz1 * sizeof(float), hipMemcpyHostToDevice) );
    checkCudaErrors(hipMemcpy(d_y1, y1, sz1 * sizeof(float), hipMemcpyHostToDevice) );
    checkCudaErrors(hipMemcpy(d_z1, z1, sz1 * sizeof(float), hipMemcpyHostToDevice) );

    cuda_interp_3_kernel<<< gridIM, block >>>(      d_interp, d_input,
                                                    d_x1, d_y1, d_z1,
                                                    m, n, o, sz1,
                                                    m2, n2, o2, sz2,
                                                    flag );
    // cuda_interp_3_kernel_tex<<< gridIM, block >>>( d_interp,
    //                                                 d_x1, d_y1, d_z1,
    //                                                 m, n, o,
    //                                                 m2, n2, o2,
    //                                                 flag,
    //                                                 texData );
    checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors( hipMemcpy( interp, d_interp, sz1 * sizeof(float), hipMemcpyDeviceToHost ) );

    // checkCudaErrors(hipDestroyTextureObject(texData));
    // checkCudaErrors(hipFreeArray(d_input3DArray));
	checkCudaErrors(hipFree(d_x1));
	checkCudaErrors(hipFree(d_y1));
	checkCudaErrors(hipFree(d_z1));
	checkCudaErrors(hipFree(d_interp));
	checkCudaErrors(hipFree(d_input));
}

extern "C" void
cuda_filter1( float *output,
              float *input,
              int m, int n, int o,
              float *filter,
              int length,
              int hw,
              int dim )
{
    //hipDeviceReset();

    if (false)
    {
        size_t freeMem, totalMem;
        checkCudaErrors(hipMemGetInfo(&freeMem,&totalMem));
        printf("\n ||| CUDA_FILTER1 : Device - Initial Free Memory: %lu / %lu ||| \n",freeMem,totalMem);
    }

    hipDeviceProp_t devProp;
    hipGetDeviceProperties( &devProp, 0 );
    //dim3 block( devProp.maxThreadsPerBlock );
    dim3 block( THREADS );

    int sz1=m*n*o;
    int3 grid = cuda_find_grid( sz1, block, devProp );
    dim3 gridIM( grid.x, grid.y, grid.z );

    float *d_output, *d_input;
    checkCudaErrors(hipMalloc( (void**) &d_output, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMemset( d_output, 0, sz1 * sizeof(float) ) );

    checkCudaErrors(hipMalloc( (void**) &d_input, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMemcpy( d_input, input, sz1 * sizeof(float), hipMemcpyHostToDevice ) );

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = d_input;
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  	resDesc.res.linear.desc.x = 32; // bits per channel
  	resDesc.res.linear.sizeInBytes = sz1 * sizeof(float);

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t texData=0;
    checkCudaErrors(hipCreateTextureObject(&texData, &resDesc, &texDesc, NULL));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL( d_filter), filter, length * sizeof(float) ) );

    // checkCudaErrors( hipBindTexture(0, texData, d_input, sz1 * sizeof(float) ) );

    cuda_filter_1_kernel<<< gridIM, block >>>( d_output, m, n, o, length, hw, dim, texData );
    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy(output, d_output, sz1 * sizeof(float), hipMemcpyDeviceToHost ) );

    checkCudaErrors(hipDestroyTextureObject(texData));
    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
}

extern "C" void
cuda_upsample2( float *u1, float *v1, float *w1,
                float *u0, float *v0, float *w0,
                int m, int n, int o,
                int m2, int n2, int o2 )
{
    //hipDeviceReset();

    if (false)
    {
        size_t freeMem, totalMem;
        checkCudaErrors(hipMemGetInfo(&freeMem,&totalMem));
        printf("\n ||| CUDA_UPSAMP2 : Device - Initial Free Memory: %lu / %lu ||| \n",freeMem,totalMem);
    }

    hipDeviceProp_t devProp;
    hipGetDeviceProperties( &devProp, 0 );
    //dim3 block( devProp.maxThreadsPerBlock );
    dim3 block( THREADS );

    int sz1=m*n*o;
    int sz2=m2*n2*o2;
    int3 grid = cuda_find_grid( sz1, block, devProp );
    dim3 gridIM( grid.x, grid.y, grid.z );

    float *d_interp, *d_input, *d_x1, *d_y1, *d_z1;

    checkCudaErrors(hipMalloc( (void**) &d_interp, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &d_input, sz2 * sizeof(float) ) );

    checkCudaErrors(hipMalloc((void **)&d_x1 , sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc((void **)&d_y1 , sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc((void **)&d_z1 , sz1 * sizeof(float) ) );

	float scale_m=(float)m/(float)m2;
	float scale_n=(float)n/(float)n2;
	float scale_o=(float)o/(float)o2;

	cuda_set_def_scale_kernel<<< gridIM, block>>>( d_x1, d_y1, d_z1, scale_m, scale_n, scale_o, m, n, o, sz1 );
	hipDeviceSynchronize();

    checkCudaErrors( hipMemcpy( d_input, u0, sz2 * sizeof(float), hipMemcpyHostToDevice ) );
    cuda_interp_3_kernel<<< gridIM, block >>>( d_interp, d_input,
                                               d_x1, d_y1, d_z1,
                                               m, n, o, sz1,
                                               m2, n2, o2, sz2,
                                               false );
    hipDeviceSynchronize();
	checkCudaErrors( hipMemcpy( u1, d_interp, sz1 * sizeof(float), hipMemcpyDeviceToHost ) );

    checkCudaErrors( hipMemcpy( d_input, v0, sz2 * sizeof(float), hipMemcpyHostToDevice ) );
    cuda_interp_3_kernel<<< gridIM, block >>>( d_interp, d_input,
                                               d_x1, d_y1, d_z1,
                                               m, n, o, sz1,
                                               m2, n2, o2, sz2,
                                               false );
    hipDeviceSynchronize();
	checkCudaErrors( hipMemcpy( v1, d_interp, sz1 * sizeof(float), hipMemcpyDeviceToHost ) );

    checkCudaErrors( hipMemcpy( d_input, w0, sz2 * sizeof(float), hipMemcpyHostToDevice ) );
    cuda_interp_3_kernel<<< gridIM, block >>>( d_interp, d_input,
                                               d_x1, d_y1, d_z1,
                                               m, n, o, sz1,
                                               m2, n2, o2, sz2,
                                               false );
    hipDeviceSynchronize();
	checkCudaErrors( hipMemcpy( w1, d_interp, sz1 * sizeof(float), hipMemcpyDeviceToHost ) );

	checkCudaErrors(hipFree(d_x1));
	checkCudaErrors(hipFree(d_y1));
	checkCudaErrors(hipFree(d_z1));
	checkCudaErrors(hipFree(d_interp));
	checkCudaErrors(hipFree(d_input));
}


struct is_positive
{
    __host__ __device__
    bool operator()(const float &a) const
    {
        return (a > 0.f);
    }
};
struct is_negative
{
    __host__ __device__
    bool operator()(const float &a) const
    {
        return (a < 0.f);
    }
};

extern "C" float
cuda_jacobian( float *u, float *v, float *w, float *grad,
               int m, int n, int o, int factor )
{
    //hipDeviceReset();

    if (false)
    {
        size_t freeMem, totalMem;
        checkCudaErrors(hipMemGetInfo(&freeMem,&totalMem));
        printf("\n ||| Before CUDA_JACOBIAN : Device - Initial Free Memory: %lu / %lu ||| \n",freeMem,totalMem);
    }

    hipDeviceProp_t devProp;
    hipGetDeviceProperties( &devProp, 0 );
    dim3 block( devProp.maxThreadsPerBlock );
    // dim3 block( THREADS );

    int sz1=m*n*o;
    int3 grid = cuda_find_grid( sz1, block, devProp );
    dim3 gridIM( grid.x, grid.y, grid.z );

	float factor1=1.0/(float)factor;

    float *d_j;
    checkCudaErrors( hipMalloc( (void**)&d_j, sz1 * sizeof(float) ) );
    checkCudaErrors( hipMemset( d_j, 0.0f, sz1 * sizeof(float) ) );

    float *j11, *j12, *j13;
    checkCudaErrors(hipMalloc( (void**) &j11, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &j12, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &j13, sz1 * sizeof(float) ) );
    float *j21, *j22, *j23;
    checkCudaErrors(hipMalloc( (void**) &j21, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &j22, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &j23, sz1 * sizeof(float) ) );
    float *j31, *j32, *j33;
    checkCudaErrors(hipMalloc( (void**) &j31, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &j32, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &j33, sz1 * sizeof(float) ) );

    float *d_input;
    checkCudaErrors(hipMalloc( (void**) &d_input, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL( d_filter), grad, 3 * sizeof(float) ) );

    checkCudaErrors(hipMemcpy( d_input, u, sz1 * sizeof(float), hipMemcpyHostToDevice ) );
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = d_input;
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  	resDesc.res.linear.desc.x = 32; // bits per channel
  	resDesc.res.linear.sizeInBytes = sz1 * sizeof(float);

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t texData=0;
    checkCudaErrors(hipCreateTextureObject(&texData, &resDesc, &texDesc, NULL));

    cuda_filter_1_kernel<<< gridIM, block >>>( j11, m, n, o, 3, 1, 2, texData );
    // hipDeviceSynchronize();
    cuda_filter_1_kernel<<< gridIM, block >>>( j12, m, n, o, 3, 1, 1, texData );
    // hipDeviceSynchronize();
    cuda_filter_1_kernel<<< gridIM, block >>>( j13, m, n, o, 3, 1, 3, texData );
    // hipDeviceSynchronize();

    checkCudaErrors(hipDestroyTextureObject(texData));
    checkCudaErrors(hipMemcpy( d_input, v, sz1 * sizeof(float), hipMemcpyHostToDevice ) );
    checkCudaErrors(hipCreateTextureObject(&texData, &resDesc, &texDesc, NULL));

    cuda_filter_1_kernel<<< gridIM, block >>>( j21, m, n, o, 3, 1, 2, texData );
    // hipDeviceSynchronize();
    cuda_filter_1_kernel<<< gridIM, block >>>( j22, m, n, o, 3, 1, 1, texData );
    // hipDeviceSynchronize();
    cuda_filter_1_kernel<<< gridIM, block >>>( j23, m, n, o, 3, 1, 3, texData );
    // hipDeviceSynchronize();

    checkCudaErrors(hipDestroyTextureObject(texData));
    checkCudaErrors(hipMemcpy( d_input, w, sz1 * sizeof(float), hipMemcpyHostToDevice ) );
    checkCudaErrors(hipCreateTextureObject(&texData, &resDesc, &texDesc, NULL));

    cuda_filter_1_kernel<<< gridIM, block >>>( j31, m, n, o, 3, 1, 2, texData );
    // hipDeviceSynchronize();
    cuda_filter_1_kernel<<< gridIM, block >>>( j32, m, n, o, 3, 1, 1, texData );
    // hipDeviceSynchronize();
    cuda_filter_1_kernel<<< gridIM, block >>>( j33, m, n, o, 3, 1, 3, texData );
    hipDeviceSynchronize();

    checkCudaErrors(hipDestroyTextureObject(texData));
    checkCudaErrors(hipFree(d_input));

    cuda_jacobian_determinant_kernel<<< gridIM, block >>>( j11, j12, j13, j21, j22, j23, j31, j32, j33, d_j, factor1, sz1 );
    hipDeviceSynchronize();

    checkCudaErrors(hipFree(j11));
    checkCudaErrors(hipFree(j12));
    checkCudaErrors(hipFree(j13));
    checkCudaErrors(hipFree(j21));
    checkCudaErrors(hipFree(j22));
    checkCudaErrors(hipFree(j23));
    checkCudaErrors(hipFree(j31));
    checkCudaErrors(hipFree(j32));
    checkCudaErrors(hipFree(j33));

    thrust::device_ptr<float> j_ptr( (float*)d_j );

	float jmean = thrust::reduce( j_ptr, j_ptr+sz1, 0.f, thrust::plus<float>() );
	jmean /= (float)sz1;

	float Jmin = thrust::reduce( j_ptr, j_ptr+sz1, 999.f, thrust::minimum<float>() );
	float Jmax = thrust::reduce( j_ptr, j_ptr+sz1, 0.f, thrust::maximum<float>() );

	float neg = (float)thrust::count_if( j_ptr, j_ptr+sz1, is_negative() );
	//float counter = (float)thrust::count_if( j_ptr, j_ptr+sz1, is_positive() );

    cuda_pow<<< gridIM, block >>>( d_j, jmean, 2.f, sz1 );
    hipDeviceSynchronize();
	float jstd = thrust::reduce( j_ptr, j_ptr+sz1, 0.f, thrust::plus<float>() );
	jstd /= (float)(sz1-1);
	jstd = sqrt(jstd);

	//float frac = neg / counter;
	float frac = neg / (float)sz1;

	std::cout<<"Jacobian of deformations| Mean (std): "<<round(jmean*1000)/1000.0<<" ("<<round(jstd*1000)/1000.0<<")\n";
	std::cout<<"Range: ["<<Jmin<<", "<<Jmax<<"] Negative fraction: "<<frac<<"\n";

    checkCudaErrors(hipFree(d_j));

    if (false)
    {
        size_t freeMem, totalMem;
        checkCudaErrors(hipMemGetInfo(&freeMem,&totalMem));
        printf("\n ||| After CUDA_JACOBIAN : Device - Initial Free Memory: %lu / %lu ||| \n",freeMem,totalMem);
    }

    return jstd;
}


extern "C" void
cuda_consistentMapping( float *u, float *v, float *w,
                        float *u2, float *v2, float *w2,
                        int m, int n, int o, int factor )
{
    //hipDeviceReset();

    if (false)
    {
        size_t freeMem, totalMem;
        checkCudaErrors(hipMemGetInfo(&freeMem,&totalMem));
        printf("\n ||| CUDA_COMBODEF : Device - Initial Free Memory: %lu / %lu ||| \n",freeMem,totalMem);
    }

    hipDeviceProp_t devProp;
    hipGetDeviceProperties( &devProp, 0 );
    //dim3 block( devProp.maxThreadsPerBlock );
    dim3 block( THREADS );

    int sz1 = m*n*o;
    int3 grid = cuda_find_grid( sz1, block, devProp );
    dim3 gridIM( grid.x, grid.y, grid.z );

	float factor1=1.0f/(float)factor;

	float *d_u, *d_v, *d_w;
    checkCudaErrors(hipMalloc( (void**) &d_u, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &d_v, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &d_w, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMemcpy( d_u, u, sz1 * sizeof(float), hipMemcpyHostToDevice ) );
    checkCudaErrors(hipMemcpy( d_v, v, sz1 * sizeof(float), hipMemcpyHostToDevice ) );
    checkCudaErrors(hipMemcpy( d_w, w, sz1 * sizeof(float), hipMemcpyHostToDevice ) );

	float *d_u2, *d_v2, *d_w2;
    checkCudaErrors(hipMalloc( (void**) &d_u2, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &d_v2, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &d_w2, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMemcpy( d_u2, u2, sz1 * sizeof(float), hipMemcpyHostToDevice ) );
    checkCudaErrors(hipMemcpy( d_v2, v2, sz1 * sizeof(float), hipMemcpyHostToDevice ) );
    checkCudaErrors(hipMemcpy( d_w2, w2, sz1 * sizeof(float), hipMemcpyHostToDevice ) );

	float *d_us, *d_vs, *d_ws;
    checkCudaErrors(hipMalloc( (void**) &d_us, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &d_vs, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &d_ws, sz1 * sizeof(float) ) );

	float *d_us2, *d_vs2, *d_ws2;
    checkCudaErrors(hipMalloc( (void**) &d_us2, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &d_vs2, sz1 * sizeof(float) ) );
    checkCudaErrors(hipMalloc( (void**) &d_ws2, sz1 * sizeof(float) ) );

    cuda_mult_vect_kernel<<< gridIM, block >>>( d_us, d_u, factor1, sz1 );
    hipDeviceSynchronize();

    cuda_mult_vect_kernel<<< gridIM, block >>>( d_vs, d_v, factor1, sz1 );
    hipDeviceSynchronize();

    cuda_mult_vect_kernel<<< gridIM, block >>>( d_ws, d_w, factor1, sz1 );
    hipDeviceSynchronize();

    cuda_mult_vect_kernel<<< gridIM, block >>>( d_us2, d_u2, factor1, sz1 );
    hipDeviceSynchronize();

    cuda_mult_vect_kernel<<< gridIM, block >>>( d_vs2, d_v2, factor1, sz1 );
    hipDeviceSynchronize();

    cuda_mult_vect_kernel<<< gridIM, block >>>( d_ws2, d_w2, factor1, sz1 );
    hipDeviceSynchronize();

    for(int it=0;it<10;it++)
    {
        cuda_interp_3_kernel<<< gridIM, block >>>( d_u, d_us2,
                                                   d_us, d_vs, d_ws,
                                                   m, n, o, sz1,
                                                   m, n, o, sz1,
                                                   true );
        hipDeviceSynchronize();

        cuda_interp_3_kernel<<< gridIM, block >>>( d_v, d_vs2,
                                                   d_us, d_vs, d_ws,
                                                   m, n, o, sz1,
                                                   m, n, o, sz1,
                                                   true );
        hipDeviceSynchronize();

        cuda_interp_3_kernel<<< gridIM, block >>>( d_w, d_ws2,
                                                   d_us, d_vs, d_ws,
                                                   m, n, o, sz1,
                                                   m, n, o, sz1,
                                                   true );
        hipDeviceSynchronize();

        cuda_subtract_vectors_kernel<<< gridIM, block >>>( d_us, d_u, 0.5f, sz1 );
        hipDeviceSynchronize();
        cuda_subtract_vectors_kernel<<< gridIM, block >>>( d_vs, d_v, 0.5f, sz1 );
        hipDeviceSynchronize();
        cuda_subtract_vectors_kernel<<< gridIM, block >>>( d_ws, d_w, 0.5f, sz1 );
        hipDeviceSynchronize();

        cuda_interp_3_kernel<<< gridIM, block >>>( d_u2, d_us,
                                                   d_us2, d_vs2, d_ws2,
                                                   m, n, o, sz1,
                                                   m, n, o, sz1,
                                                   true );
        hipDeviceSynchronize();

        cuda_interp_3_kernel<<< gridIM, block >>>( d_v2, d_vs,
                                                   d_us2, d_vs2, d_ws2,
                                                   m, n, o, sz1,
                                                   m, n, o, sz1,
                                                   true );
        hipDeviceSynchronize();

        cuda_interp_3_kernel<<< gridIM, block >>>( d_w2, d_ws,
                                                   d_us2, d_vs2, d_ws2,
                                                   m, n, o, sz1,
                                                   m, n, o, sz1,
                                                   true );
        hipDeviceSynchronize();

        cuda_subtract_vectors_kernel<<< gridIM, block >>>( d_us2, d_u2, 0.5f, sz1 );
        hipDeviceSynchronize();
        cuda_subtract_vectors_kernel<<< gridIM, block >>>( d_vs2, d_v2, 0.5f, sz1 );
        hipDeviceSynchronize();
        cuda_subtract_vectors_kernel<<< gridIM, block >>>( d_ws2, d_w2, 0.5f, sz1 );
        hipDeviceSynchronize();

        checkCudaErrors(hipMemcpy( d_us, d_u, sz1 * sizeof(float), hipMemcpyDeviceToDevice ) );
        checkCudaErrors(hipMemcpy( d_vs, d_v, sz1 * sizeof(float), hipMemcpyDeviceToDevice ) );
        checkCudaErrors(hipMemcpy( d_ws, d_w, sz1 * sizeof(float), hipMemcpyDeviceToDevice ) );
        checkCudaErrors(hipMemcpy( d_us2, d_u2, sz1 * sizeof(float), hipMemcpyDeviceToDevice ) );
        checkCudaErrors(hipMemcpy( d_vs2, d_v2, sz1 * sizeof(float), hipMemcpyDeviceToDevice ) );
        checkCudaErrors(hipMemcpy( d_ws2, d_w2, sz1 * sizeof(float), hipMemcpyDeviceToDevice ) );
    }

    cuda_mult_vect_kernel<<< gridIM, block >>>( d_u, d_us, (float)factor, sz1 );
    hipDeviceSynchronize();

    cuda_mult_vect_kernel<<< gridIM, block >>>( d_v, d_vs, (float)factor, sz1 );
    hipDeviceSynchronize();

    cuda_mult_vect_kernel<<< gridIM, block >>>( d_w, d_ws, (float)factor, sz1 );
    hipDeviceSynchronize();

    cuda_mult_vect_kernel<<< gridIM, block >>>( d_u2, d_us2, (float)factor, sz1 );
    hipDeviceSynchronize();

    cuda_mult_vect_kernel<<< gridIM, block >>>( d_v2, d_vs2, (float)factor, sz1 );
    hipDeviceSynchronize();

    cuda_mult_vect_kernel<<< gridIM, block >>>( d_w2, d_ws2, (float)factor, sz1 );
    hipDeviceSynchronize();

    checkCudaErrors(hipMemcpy( u, d_u, sz1 * sizeof(float), hipMemcpyDeviceToHost ) );
    checkCudaErrors(hipMemcpy( v, d_v, sz1 * sizeof(float), hipMemcpyDeviceToHost ) );
    checkCudaErrors(hipMemcpy( w, d_w, sz1 * sizeof(float), hipMemcpyDeviceToHost ) );
    checkCudaErrors(hipMemcpy( u2, d_u2, sz1 * sizeof(float), hipMemcpyDeviceToHost ) );
    checkCudaErrors(hipMemcpy( v2, d_v2, sz1 * sizeof(float), hipMemcpyDeviceToHost ) );
    checkCudaErrors(hipMemcpy( w2, d_w2, sz1 * sizeof(float), hipMemcpyDeviceToHost ) );
    hipDeviceSynchronize();

    checkCudaErrors(hipFree(d_ws2));
    checkCudaErrors(hipFree(d_vs2));
    checkCudaErrors(hipFree(d_us2));
    checkCudaErrors(hipFree(d_ws));
    checkCudaErrors(hipFree(d_vs));
    checkCudaErrors(hipFree(d_us));
    checkCudaErrors(hipFree(d_w2));
    checkCudaErrors(hipFree(d_v2));
    checkCudaErrors(hipFree(d_u2));
    checkCudaErrors(hipFree(d_w));
    checkCudaErrors(hipFree(d_v));
    checkCudaErrors(hipFree(d_u));
}
